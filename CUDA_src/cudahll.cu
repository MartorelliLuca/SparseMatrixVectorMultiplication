#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <string.h>
#include <errno.h>
#include <dirent.h>

#include "../CUDA_include/cudahll.h"
#include "../src/data_structures/hll_matrix.h"
#include "../src/data_structures/performance.h"
#include "../CUDA_include/hll/cuda_hll_kernel_v1.cuh"

float invoke_kernel_1(HLL_matrix *hll_matrix, double *x, double *z)
{
    int thread_num = 256;
    int block_num = hll_matrix->hacks_num / thread_num;
    double *data;
    double *d_x, *d_y;
    int *col_indexes, *maxnrz, *offsets;
    float time;

    hipError_t error;
    hipEvent_t start, stop;

    // Memory allocation for Device
    error = hipMalloc(&data, hll_matrix->data_num * sizeof(double));
    if (error != hipSuccess)
    {
        printf("Error occour in hipMalloc in invoke kernel 1 for data\nError: %s\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    error = hipMalloc(&col_indexes, hll_matrix->data_num * sizeof(int));
    if (error != hipSuccess)
    {
        printf("Error occour in hipMalloc in invoke kernel 1 for col indexes\nError: %s\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    error = hipMalloc(&maxnrz, hll_matrix->hacks_num * sizeof(int));
    if (error != hipSuccess)
    {
        printf("Error occour in hipMalloc in invoke kernel 1 for maxnrz\nError: %s\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    error = hipMalloc(&offsets, hll_matrix->offsets_num * sizeof(int));
    if (error != hipSuccess)
    {
        printf("Error occour in hipMalloc in invoke kernel 1 for offsets\nError: %s\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    error = hipMalloc(&d_x, hll_matrix->data_num * sizeof(int));
    if (error != hipSuccess)
    {
        printf("Error occour in hipMalloc in invoke kernel 1 for d_x\nError: %s\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    error = hipMalloc(&d_y, hll_matrix->data_num * sizeof(int));
    if (error != hipSuccess)
    {
        printf("Error occour in hipMalloc in invoke kernel 1 for d_y\nError: %s\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    // Copy data from host to device
    error = hipMemcpy(data, hll_matrix->data, hll_matrix->data_num, hipMemcpyHostToDevice);
    if (error != hipSuccess)
    {
        printf("Error occour in hipMemcpy in invoke kernel 1 for data\nError: %s\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    error = hipMemcpy(col_indexes, hll_matrix->col_index, hll_matrix->data_num, hipMemcpyHostToDevice);
    if (error != hipSuccess)
    {
        printf("Error occour in hipMemcpy in invoke kernel 1 for col indexes\nError: %s\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    error = hipMemcpy(maxnrz, hll_matrix->max_nzr, hll_matrix->hacks_num, hipMemcpyHostToDevice);
    if (error != hipSuccess)
    {
        printf("Error occour in hipMemcpy in invoke kernel 1 for col indexes\nError: %s\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    error = hipMemcpy(offsets, hll_matrix->offsets, hll_matrix->offsets_num, hipMemcpyHostToDevice);
    if (error != hipSuccess)
    {
        printf("Error occour in hipMemcpy in invoke kernel 1 for col indexes\nError: %s\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    error = hipMemcpy(d_x, x, hll_matrix->offsets_num, hipMemcpyHostToDevice);
    if (error != hipSuccess)
    {
        printf("Error occour in hipMemcpy in invoke kernel 1 for col indexes\nError: %s\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    // Configurazione dei thread e lancio del kernel

    // Eventi CUDA per la misurazione del tempo
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);
    // Invocazione del Kernel CUDA
    cuda_kernel_0<<<block_num, thread_num>>>(d_y, hll_matrix->hack_size, hll_matrix->hacks_num, data, offsets, col_indexes, maxnrz, d_x, hll_matrix->N);
    hipEventRecord(stop, 0);

    hipEventSynchronize(stop);

    // Calculation of execution time in milliseconds
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    // Copy results from device to host
    error = hipMemcpy(z, d_y, hll_matrix->offsets_num, hipMemcpyDeviceToHost);
    if (error != hipSuccess)
    {
        printf("Error occour in hipMemcpy in invoke kernel 1 for col d_y\nError: %s\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    // Calculation of performance
    time = milliseconds / 1000;

    // Device memory deallocation and CUDA events
    hipFree(data);
    hipFree(col_indexes);
    hipFree(maxnrz);
    hipFree(offsets);
    hipFree(d_y);
    hipFree(d_x);

    error = hipDeviceReset();
    if (error != hipSuccess)
    {
        printf("Error occour in hipDeviceReset in invoke kernel 1\nError: %s\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return time;
}
