#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <string.h>
#include <errno.h>
#include <dirent.h>

#include "../CUDA_include/cudahll.h"
#include "../src/data_structures/hll_matrix.h"
#include "../src/data_structures/performance.h"
#include "../CUDA_include/hll/cuda_hll_kernel_v1.cuh"
#include "../CUDA_include/hll/cuda_hll_kernel_v2.cuh"

float invoke_kernel_1(HLL_matrix *hll_matrix, double *x, double *z, int num_threads)
{
    int block_num = (hll_matrix->hacks_num + num_threads - 1) / num_threads;

    double *d_data, *d_x, *d_y;
    int *d_col_index, *d_max_nzr, *d_offsets;
    hipEvent_t start, stop;
    hipError_t error;

    error = hipMalloc(&d_data, hll_matrix->data_num * sizeof(double));
    if (error != hipSuccess)
    {
        printf("Error occour in cuda malloc for d_data in invoke kernel 1\nError: %s\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    error = hipMalloc(&d_col_index, hll_matrix->data_num * sizeof(int));
    if (error != hipSuccess)
    {
        printf("Error occour in cuda malloc for d_col_index in invoke kernel 1\nError: %s\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    error = hipMalloc(&d_max_nzr, hll_matrix->hacks_num * sizeof(int));
    if (error != hipSuccess)
    {
        printf("Error occour in cuda malloc for d_max_nzr in invoke kernel 1\nError: %s\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    error = hipMalloc(&d_offsets, hll_matrix->offsets_num * sizeof(int));
    if (error != hipSuccess)
    {
        printf("Error occour in cuda malloc for d_offsets in invoke kernel 1\nError: %s\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    error = hipMalloc(&d_x, hll_matrix->N * sizeof(double));
    if (error != hipSuccess)
    {
        printf("Error occour in cuda malloc for d_max_nzr in invoke kernel 1\nError: %s\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    error = hipMalloc(&d_y, hll_matrix->M * sizeof(double));
    if (error != hipSuccess)
    {
        printf("Error occour in cuda malloc for d_y in invoke kernel 1\nError: %s\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    error = hipMemcpy(d_data, hll_matrix->data, hll_matrix->data_num * sizeof(double), hipMemcpyHostToDevice);
    if (error != hipSuccess)
    {
        printf("Error occour in cuda memcpy for d_data in invoke kernel 1\nError: %s\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    error = hipMemcpy(d_col_index, hll_matrix->col_index, hll_matrix->data_num * sizeof(int), hipMemcpyHostToDevice);
    if (error != hipSuccess)
    {
        printf("Error occour in cuda memcpy for d_col_index in invoke kernel 1\nError: %s\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    error = hipMemcpy(d_max_nzr, hll_matrix->max_nzr, hll_matrix->hacks_num * sizeof(int), hipMemcpyHostToDevice);
    if (error != hipSuccess)
    {
        printf("Error occour in cuda memcpy for d_max_nzr in invoke kernel 1\nError: %s\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    error = hipMemcpy(d_offsets, hll_matrix->offsets, hll_matrix->offsets_num * sizeof(int), hipMemcpyHostToDevice);
    if (error != hipSuccess)
    {
        printf("Error occour in cuda memcpy for d_offsets in invoke kernel 1\nError: %s\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    error = hipMemcpy(d_x, x, hll_matrix->N * sizeof(double), hipMemcpyHostToDevice);
    if (error != hipSuccess)
    {
        printf("Error occour in cuda memcpy for d_x in invoke kernel 1\nError: %s\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    cuda_kernel_1<<<block_num, num_threads>>>(d_y, hll_matrix->hack_size, hll_matrix->hacks_num, d_data, d_offsets, d_col_index, d_max_nzr, d_x, hll_matrix->M);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    error = hipMemcpy(z, d_y, hll_matrix->M * sizeof(double), hipMemcpyDeviceToHost);
    if (error != hipSuccess)
    {
        printf("Error occour in cuda memcpy for d_y in invoke kernel 1\nError: %s\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    hipFree(d_data);
    hipFree(d_col_index);
    hipFree(d_max_nzr);
    hipFree(d_offsets);
    hipFree(d_x);
    hipFree(d_y);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return milliseconds / 1000.0;
}

float invoke_kernel_2(HLL_matrix *hll_matrix, double *x, double *z, int num_threads)
{
    int M = hll_matrix->M;
    int block_num = (hll_matrix->hacks_num + num_threads - 1) / num_threads;

    double *d_data, *d_x, *d_y;
    int *d_col_index, *d_max_nzr, *d_offsets;
    hipMalloc(&d_data, hll_matrix->data_num * sizeof(double));
    hipMalloc(&d_col_index, hll_matrix->data_num * sizeof(int));
    hipMalloc(&d_max_nzr, hll_matrix->hacks_num * sizeof(int));
    hipMalloc(&d_offsets, hll_matrix->offsets_num * sizeof(int));
    hipMalloc(&d_x, hll_matrix->N * sizeof(double));
    hipMalloc(&d_y, hll_matrix->M * sizeof(double));

    hipMemcpy(d_data, hll_matrix->data, hll_matrix->data_num * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_col_index, hll_matrix->col_index, hll_matrix->data_num * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_max_nzr, hll_matrix->max_nzr, hll_matrix->hacks_num * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_offsets, hll_matrix->offsets, hll_matrix->offsets_num * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_x, x, hll_matrix->N * sizeof(double), hipMemcpyHostToDevice);

    hipMemset(d_y, 0, hll_matrix->M * sizeof(double));

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    int shared_mem_size = num_threads * sizeof(double);
    cuda_kernel_2<<<block_num, num_threads, shared_mem_size>>>(d_y, hll_matrix->hack_size, hll_matrix->hacks_num, d_data, d_offsets, d_col_index, d_max_nzr, d_x, hll_matrix->M);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    hipMemcpy(z, d_y, hll_matrix->M * sizeof(double), hipMemcpyDeviceToHost);

    hipFree(d_data);
    hipFree(d_col_index);
    hipFree(d_max_nzr);
    hipFree(d_offsets);
    hipFree(d_x);
    hipFree(d_y);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return milliseconds / 1000.0;
}