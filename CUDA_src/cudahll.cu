#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <string.h>
#include <errno.h>
#include <dirent.h>

#include "../CUDA_include/cudahll.h"
#include "../src/data_structures/hll_matrix.h"
#include "../src/data_structures/performance.h"
#include "../CUDA_include/hll/cuda_hll_kernel_v1.cuh"
#include "../CUDA_include/hll/cuda_hll_kernel_v2.cuh"
#include "../CUDA_include/hll/cuda_hll_kernel_v3.cuh"
#include "../CUDA_include/hll/cuda_hll_kernel_v4.cuh"

#define WARP_SIZE 32

#define CHECK_CUDA(call) check_cuda((call), __FILE__, __LINE__)

void print_error(hipError_t *error, int kernel_index)
{
    if (*error != hipSuccess)
    {
        // printf("Error occour in invoke kernel %d\nError: %s\n", kernel_index, hipGetErrorString(*error));
        exit(EXIT_FAILURE);
    }
}

inline void check_cuda(hipError_t err, const char *file, int line)
{
    if (err != hipSuccess)
    {
        // fprintf(stderr, "CUDA Error: %s (file %s, line %d)\n", hipGetErrorString(err), file, line);
        exit(EXIT_FAILURE);
    }
}

float invoke_kernel_1(HLL_matrix *hll_matrix, double *x, double *z, int num_threads)
{
    dim3 block_dim(WARP_SIZE, num_threads);
    dim3 grid_dim((hll_matrix->N + block_dim.y - 1) / block_dim.y);

    double *d_data, *d_x, *d_y;
    int *d_col_index, *d_max_nzr, *d_offsets;
    hipEvent_t start, stop;

    // Memory allocation
    CHECK_CUDA(hipMalloc(&d_data, hll_matrix->data_num * sizeof(double)));
    CHECK_CUDA(hipMalloc(&d_col_index, hll_matrix->data_num * sizeof(int)));
    CHECK_CUDA(hipMalloc(&d_max_nzr, hll_matrix->hacks_num * sizeof(int)));
    CHECK_CUDA(hipMalloc(&d_offsets, hll_matrix->offsets_num * sizeof(int)));
    CHECK_CUDA(hipMalloc(&d_x, hll_matrix->N * sizeof(double)));
    CHECK_CUDA(hipMalloc(&d_y, hll_matrix->M * sizeof(double)));

    // Copying data
    CHECK_CUDA(hipMemcpy(d_data, hll_matrix->data, hll_matrix->data_num * sizeof(double), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_col_index, hll_matrix->col_index, hll_matrix->data_num * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_max_nzr, hll_matrix->max_nzr, hll_matrix->hacks_num * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_offsets, hll_matrix->offsets, hll_matrix->offsets_num * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_x, x, hll_matrix->N * sizeof(double), hipMemcpyHostToDevice));

    // Creating and starting events
    CHECK_CUDA(hipEventCreate(&start));
    CHECK_CUDA(hipEventCreate(&stop));
    CHECK_CUDA(hipEventRecord(start));

    // Kernel Execution
    cuda_hll_kernel_v1<<<grid_dim, num_threads>>>(d_y, hll_matrix->hack_size, hll_matrix->hacks_num,
                                                  d_data, d_offsets, d_col_index, d_max_nzr, d_x, hll_matrix->M);

    CHECK_CUDA(hipEventRecord(stop));
    CHECK_CUDA(hipEventSynchronize(stop));

    // Time calculation
    float milliseconds = 0;
    CHECK_CUDA(hipEventElapsedTime(&milliseconds, start, stop));

    // Copy of the result
    CHECK_CUDA(hipMemcpy(z, d_y, hll_matrix->M * sizeof(double), hipMemcpyDeviceToHost));

    // Memory Deallocation
    CHECK_CUDA(hipFree(d_data));
    CHECK_CUDA(hipFree(d_col_index));
    CHECK_CUDA(hipFree(d_max_nzr));
    CHECK_CUDA(hipFree(d_offsets));
    CHECK_CUDA(hipFree(d_x));
    CHECK_CUDA(hipFree(d_y));

    // Destruction of events
    CHECK_CUDA(hipEventDestroy(start));
    CHECK_CUDA(hipEventDestroy(stop));

    return milliseconds / 1000.0;
}

float invoke_kernel_2(HLL_matrix *hll_matrix, double *x, double *z, int num_threads)
{
    dim3 block_dim(WARP_SIZE, num_threads);
    dim3 grid_dim((hll_matrix->N + block_dim.y - 1) / block_dim.y);

    double *d_data, *d_x, *d_y;
    int *d_col_index, *d_max_nzr, *d_offsets;
    hipEvent_t start, stop;

    // Memory allocation
    CHECK_CUDA(hipMalloc(&d_data, hll_matrix->data_num * sizeof(double)));
    CHECK_CUDA(hipMalloc(&d_col_index, hll_matrix->data_num * sizeof(int)));
    CHECK_CUDA(hipMalloc(&d_max_nzr, hll_matrix->hacks_num * sizeof(int)));
    CHECK_CUDA(hipMalloc(&d_offsets, hll_matrix->offsets_num * sizeof(int)));
    CHECK_CUDA(hipMalloc(&d_x, hll_matrix->N * sizeof(double)));
    CHECK_CUDA(hipMalloc(&d_y, hll_matrix->N * sizeof(double)));

    // Copying data
    CHECK_CUDA(hipMemcpy(d_data, hll_matrix->data, hll_matrix->data_num * sizeof(double), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_col_index, hll_matrix->col_index, hll_matrix->data_num * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_max_nzr, hll_matrix->max_nzr, hll_matrix->hacks_num * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_offsets, hll_matrix->offsets, hll_matrix->offsets_num * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_x, x, hll_matrix->N * sizeof(double), hipMemcpyHostToDevice));

    // Creating and starting events
    CHECK_CUDA(hipEventCreate(&start));
    CHECK_CUDA(hipEventCreate(&stop));
    CHECK_CUDA(hipEventRecord(start));

    // Kernel Execution
    cuda_hll_kernel_v2<<<grid_dim, num_threads>>>(d_y, hll_matrix->hack_size, hll_matrix->hacks_num, d_data, d_offsets,
                                                  d_col_index, d_max_nzr, d_x, hll_matrix->M);

    CHECK_CUDA(hipEventRecord(stop));
    CHECK_CUDA(hipEventSynchronize(stop));

    // Time calculation
    float milliseconds = 0;
    CHECK_CUDA(hipEventElapsedTime(&milliseconds, start, stop));

    // Copy of the result
    CHECK_CUDA(hipMemcpy(z, d_y, hll_matrix->M * sizeof(double), hipMemcpyDeviceToHost));

    // Memory Deallocation
    CHECK_CUDA(hipFree(d_data));
    CHECK_CUDA(hipFree(d_col_index));
    CHECK_CUDA(hipFree(d_max_nzr));
    CHECK_CUDA(hipFree(d_offsets));
    CHECK_CUDA(hipFree(d_x));
    CHECK_CUDA(hipFree(d_y));

    // Destruction of events
    CHECK_CUDA(hipEventDestroy(start));
    CHECK_CUDA(hipEventDestroy(stop));

    return milliseconds / 1000.0;
}

float invoke_kernel_3(HLL_matrix *hll_matrix, double *x, double *z, int num_threads)
{
    dim3 block_dim(WARP_SIZE, num_threads);
    dim3 grid_dim((hll_matrix->N + block_dim.y - 1) / block_dim.y);

    double *d_data, *d_x, *d_y;
    int *d_col_index, *d_max_nzr, *d_offsets;
    hipEvent_t start, stop;

    // Memory allocation
    CHECK_CUDA(hipMalloc(&d_data, hll_matrix->data_num * sizeof(double)));
    CHECK_CUDA(hipMalloc(&d_col_index, hll_matrix->data_num * sizeof(int)));
    CHECK_CUDA(hipMalloc(&d_max_nzr, hll_matrix->hacks_num * sizeof(int)));
    CHECK_CUDA(hipMalloc(&d_offsets, hll_matrix->offsets_num * sizeof(int)));
    CHECK_CUDA(hipMalloc(&d_x, hll_matrix->N * sizeof(double)));
    CHECK_CUDA(hipMalloc(&d_y, hll_matrix->N * sizeof(double)));

    // Copying data
    CHECK_CUDA(hipMemcpy(d_data, hll_matrix->data, hll_matrix->data_num * sizeof(double), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_col_index, hll_matrix->col_index, hll_matrix->data_num * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_max_nzr, hll_matrix->max_nzr, hll_matrix->hacks_num * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_offsets, hll_matrix->offsets, hll_matrix->offsets_num * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_x, x, hll_matrix->N * sizeof(double), hipMemcpyHostToDevice));

    // Creating and starting events
    CHECK_CUDA(hipEventCreate(&start));
    CHECK_CUDA(hipEventCreate(&stop));
    CHECK_CUDA(hipEventRecord(start));

    // Kernel Execution
    int shared_mem_size = 1024 * sizeof(double);
    cuda_hll_kernel_v3<<<grid_dim, num_threads, shared_mem_size>>>(d_y, hll_matrix->hack_size, hll_matrix->hacks_num,
                                                                   d_data, d_offsets, d_col_index, d_max_nzr, d_x,
                                                                   hll_matrix->M);

    CHECK_CUDA(hipEventRecord(stop));
    CHECK_CUDA(hipEventSynchronize(stop));

    // Time calculation
    float milliseconds = 0;
    CHECK_CUDA(hipEventElapsedTime(&milliseconds, start, stop));

    // Copy of the result
    CHECK_CUDA(hipMemcpy(z, d_y, hll_matrix->N * sizeof(double), hipMemcpyDeviceToHost));

    // Memory Deallocation
    CHECK_CUDA(hipFree(d_data));
    CHECK_CUDA(hipFree(d_col_index));
    CHECK_CUDA(hipFree(d_max_nzr));
    CHECK_CUDA(hipFree(d_offsets));
    CHECK_CUDA(hipFree(d_x));
    CHECK_CUDA(hipFree(d_y));

    // Destruction of events
    CHECK_CUDA(hipEventDestroy(start));
    CHECK_CUDA(hipEventDestroy(stop));

    return milliseconds / 1000.0;
}

float invoke_kernel_4(HLL_matrix *hll_matrix, double *x, double *z, int num_threads)
{
    int block_num = (hll_matrix->N * 32) / num_threads;

    double *d_data, *d_x, *d_y;
    int *d_col_index, *d_max_nzr, *d_offsets;
    hipEvent_t start, stop;

    // Memory allocation
    CHECK_CUDA(hipMalloc(&d_data, hll_matrix->data_num * sizeof(double)));
    CHECK_CUDA(hipMalloc(&d_col_index, hll_matrix->data_num * sizeof(int)));
    CHECK_CUDA(hipMalloc(&d_max_nzr, hll_matrix->hacks_num * sizeof(int)));
    CHECK_CUDA(hipMalloc(&d_offsets, hll_matrix->offsets_num * sizeof(int)));
    CHECK_CUDA(hipMalloc(&d_x, hll_matrix->N * sizeof(double)));
    CHECK_CUDA(hipMalloc(&d_y, hll_matrix->N * sizeof(double)));

    // Copying data
    CHECK_CUDA(hipMemcpy(d_data, hll_matrix->data, hll_matrix->data_num * sizeof(double), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_col_index, hll_matrix->col_index, hll_matrix->data_num * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_max_nzr, hll_matrix->max_nzr, hll_matrix->hacks_num * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_offsets, hll_matrix->offsets, hll_matrix->offsets_num * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_x, x, hll_matrix->N * sizeof(double), hipMemcpyHostToDevice));

    // Creating and starting events
    CHECK_CUDA(hipEventCreate(&start));
    CHECK_CUDA(hipEventCreate(&stop));
    CHECK_CUDA(hipEventRecord(start));

    // Kernel Execution
    cuda_hll_kernel_v4<<<block_num, num_threads>>>(d_y, hll_matrix->hack_size, hll_matrix->hacks_num,
                                                   d_data, d_offsets, d_col_index, d_max_nzr, d_x,
                                                   hll_matrix->M);

    CHECK_CUDA(hipEventRecord(stop));
    CHECK_CUDA(hipEventSynchronize(stop));

    // Time calculation
    float milliseconds = 0;
    CHECK_CUDA(hipEventElapsedTime(&milliseconds, start, stop));

    // Copy of the result
    CHECK_CUDA(hipMemcpy(z, d_y, hll_matrix->N * sizeof(double), hipMemcpyDeviceToHost));

    // Memory Deallocation
    CHECK_CUDA(hipFree(d_data));
    CHECK_CUDA(hipFree(d_col_index));
    CHECK_CUDA(hipFree(d_max_nzr));
    CHECK_CUDA(hipFree(d_offsets));
    CHECK_CUDA(hipFree(d_x));
    CHECK_CUDA(hipFree(d_y));

    // Destruction of events
    CHECK_CUDA(hipEventDestroy(start));
    CHECK_CUDA(hipEventDestroy(stop));

    return milliseconds / 1000.0;
}
