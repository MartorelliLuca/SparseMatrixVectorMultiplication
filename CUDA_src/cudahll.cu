#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <string.h>
#include <errno.h>
#include <dirent.h>

#include "../CUDA_include/cudahll.h"
#include "../src/data_structures/hll_matrix.h"
#include "../src/data_structures/performance.h"
#include "../CUDA_include/cuda_hll_kernel_v1.cuh"

float invoke_kernel_1(HLL_matrix *hll_matrix, double *x, double *z)
{
    HLL_matrix d_A;
    double *d_x, *d_y;
    float time;

    // Allocazione memoria su device
    hipMalloc(&d_A.offsets, hll_matrix->offsets_num * sizeof(int));
    hipMalloc(&d_A.col_index, hll_matrix->data_num * sizeof(int));
    hipMalloc(&d_A.data, hll_matrix->data_num * sizeof(double));
    hipMalloc(&d_A.max_nzr, hll_matrix->hacks_num * sizeof(int));

    hipMalloc(&d_x, hll_matrix->N * sizeof(double));
    hipMalloc(&d_y, hll_matrix->M * sizeof(double));

    // Copia dei dati da host a device
    hipMemcpy(d_A.offsets, hll_matrix->offsets, hll_matrix->offsets_num * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_A.col_index, hll_matrix->col_index, hll_matrix->data_num * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_A.data, hll_matrix->data, hll_matrix->data_num * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_A.max_nzr, hll_matrix->max_nzr, hll_matrix->hacks_num * sizeof(int), hipMemcpyHostToDevice);

    hipMemcpy(d_x, x, hll_matrix->N * sizeof(double), hipMemcpyHostToDevice);

    // Configurazione dei thread e lancio del kernel
    int blockSize = 256;
    int gridSize = (hll_matrix->M + blockSize - 1) / blockSize;

    // Eventi CUDA per la misurazione del tempo
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start); // Inizio misurazione
    hll_matvec_kernel<<<gridSize, blockSize>>>(d_A, d_x, d_y);
    hipEventRecord(stop); // Fine misurazione

    hipEventSynchronize(stop);

    // Calcolo del tempo di esecuzione in millisecondi
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    // Copia del risultato dal device all'host
    hipMemcpy(z, d_y, hll_matrix->M * sizeof(double), hipMemcpyDeviceToHost);

    printf("Milliseconds = %.16lf\n", milliseconds);

    // Calcolo delle prestazioni in MFLOPS
    time = milliseconds / 1000.0;

    // Deallocazione memoria device e eventi CUDA
    hipFree(d_A.offsets);
    hipFree(d_A.col_index);
    hipFree(d_A.data);
    hipFree(d_A.max_nzr);
    hipFree(d_x);
    hipFree(d_y);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return time;
}
