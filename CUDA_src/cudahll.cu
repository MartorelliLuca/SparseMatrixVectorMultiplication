#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <string.h>
#include <errno.h>
#include <dirent.h>

#include "../CUDA_include/cudahll.h"
#include "../src/data_structures/hll_matrix.h"
#include "../src/data_structures/performance.h"
#include "../CUDA_include/hll/cuda_hll_kernel_v1.cuh"
#include "../CUDA_include/hll/cuda_hll_kernel_v2.cuh"
#include "../CUDA_include/hll/cuda_hll_kernel_v3.cuh"

void print_error(hipError_t *error, int kernel_index)
{
    if (*error != hipSuccess)
    {
        printf("Error occour in invoke kernel %d\nError: %s\n", kernel_index, hipGetErrorString(*error));
        exit(EXIT_FAILURE);
    }
}

float invoke_kernel_1(HLL_matrix *hll_matrix, double *x, double *z, int num_threads)
{
    int block_num = (hll_matrix->hacks_num + num_threads - 1) / num_threads;

    double *d_data, *d_x, *d_y;
    int *d_col_index, *d_max_nzr, *d_offsets;
    hipEvent_t start, stop;
    hipError_t error;

    error = hipMalloc(&d_data, hll_matrix->data_num * sizeof(double));
    print_error(&error, 1);

    error = hipMalloc(&d_col_index, hll_matrix->data_num * sizeof(int));
    print_error(&error, 1);

    error = hipMalloc(&d_max_nzr, hll_matrix->hacks_num * sizeof(int));
    print_error(&error, 1);

    error = hipMalloc(&d_offsets, hll_matrix->offsets_num * sizeof(int));
    print_error(&error, 1);

    error = hipMalloc(&d_x, hll_matrix->N * sizeof(double));
    print_error(&error, 1);

    error = hipMalloc(&d_y, hll_matrix->M * sizeof(double));
    print_error(&error, 1);

    error = hipMemcpy(d_data, hll_matrix->data, hll_matrix->data_num * sizeof(double), hipMemcpyHostToDevice);
    print_error(&error, 1);

    error = hipMemcpy(d_col_index, hll_matrix->col_index, hll_matrix->data_num * sizeof(int), hipMemcpyHostToDevice);
    print_error(&error, 1);

    error = hipMemcpy(d_max_nzr, hll_matrix->max_nzr, hll_matrix->hacks_num * sizeof(int), hipMemcpyHostToDevice);
    print_error(&error, 1);

    error = hipMemcpy(d_offsets, hll_matrix->offsets, hll_matrix->offsets_num * sizeof(int), hipMemcpyHostToDevice);
    print_error(&error, 1);

    error = hipMemcpy(d_x, x, hll_matrix->N * sizeof(double), hipMemcpyHostToDevice);
    print_error(&error, 1);

    error = hipEventCreate(&start);
    print_error(&error, 1);

    error = hipEventCreate(&stop);
    print_error(&error, 1);

    error = hipEventRecord(start);
    print_error(&error, 1);

    cuda_hll_kernel_v1<<<block_num, num_threads>>>(d_y, hll_matrix->hack_size, hll_matrix->hacks_num, d_data, d_offsets, d_col_index, d_max_nzr, d_x, hll_matrix->M);

    error = hipEventRecord(stop);
    print_error(&error, 1);

    error = hipEventSynchronize(stop);
    print_error(&error, 1);

    float milliseconds = 0;
    error = hipEventElapsedTime(&milliseconds, start, stop);
    print_error(&error, 1);

    error = hipMemcpy(z, d_y, hll_matrix->M * sizeof(double), hipMemcpyDeviceToHost);
    print_error(&error, 1);

    error = hipFree(d_data);
    print_error(&error, 1);

    error = hipFree(d_col_index);
    print_error(&error, 1);

    error = hipFree(d_max_nzr);
    print_error(&error, 1);

    error = hipFree(d_offsets);
    print_error(&error, 1);

    error = hipFree(d_x);
    print_error(&error, 1);

    error = hipFree(d_y);
    print_error(&error, 1);

    error = hipEventDestroy(start);
    print_error(&error, 1);

    error = hipEventDestroy(stop);
    print_error(&error, 1);

    return milliseconds / 1000.0;
}

float invoke_kernel_2(HLL_matrix *hll_matrix, double *x, double *z, int num_threads)
{
    int block_num = (hll_matrix->N * 32) / (double)num_threads;

    double *d_data, *d_x, *d_y;
    int *d_col_index, *d_max_nzr, *d_offsets;
    hipEvent_t start, stop;
    hipError_t error;

    error = hipMalloc(&d_data, hll_matrix->data_num * sizeof(double));
    print_error(&error, 2);

    error = hipMalloc(&d_col_index, hll_matrix->data_num * sizeof(int));
    print_error(&error, 2);

    error = hipMalloc(&d_max_nzr, hll_matrix->hacks_num * sizeof(int));
    print_error(&error, 2);

    error = hipMalloc(&d_offsets, hll_matrix->offsets_num * sizeof(int));
    print_error(&error, 2);

    error = hipMalloc(&d_x, hll_matrix->N * sizeof(double));
    print_error(&error, 2);

    error = hipMalloc(&d_y, hll_matrix->M * sizeof(double));
    print_error(&error, 2);

    error = hipMemcpy(d_data, hll_matrix->data, hll_matrix->data_num * sizeof(double), hipMemcpyHostToDevice);
    print_error(&error, 2);

    error = hipMemcpy(d_col_index, hll_matrix->col_index, hll_matrix->data_num * sizeof(int), hipMemcpyHostToDevice);
    print_error(&error, 2);

    error = hipMemcpy(d_max_nzr, hll_matrix->max_nzr, hll_matrix->hacks_num * sizeof(int), hipMemcpyHostToDevice);
    print_error(&error, 2);

    error = hipMemcpy(d_offsets, hll_matrix->offsets, hll_matrix->offsets_num * sizeof(int), hipMemcpyHostToDevice);
    print_error(&error, 2);

    error = hipMemcpy(d_x, x, hll_matrix->N * sizeof(double), hipMemcpyHostToDevice);
    print_error(&error, 2);

    error = hipEventCreate(&start);
    print_error(&error, 2);

    error = hipEventCreate(&stop);
    print_error(&error, 2);

    error = hipEventRecord(start);
    print_error(&error, 2);

    cuda_hll_kernel_v2<<<block_num, num_threads>>>(d_y, hll_matrix->hack_size, hll_matrix->hacks_num, d_data, d_offsets, d_col_index, d_max_nzr, d_x, hll_matrix->M);

    error = hipEventRecord(stop);
    print_error(&error, 2);

    error = hipEventSynchronize(stop);
    print_error(&error, 2);

    float milliseconds = 0;
    error = hipEventElapsedTime(&milliseconds, start, stop);
    print_error(&error, 2);

    error = hipMemcpy(z, d_y, hll_matrix->M * sizeof(double), hipMemcpyDeviceToHost);
    print_error(&error, 2);

    error = hipFree(d_data);
    print_error(&error, 2);

    error = hipFree(d_col_index);
    print_error(&error, 2);

    error = hipFree(d_max_nzr);
    print_error(&error, 2);

    error = hipFree(d_offsets);
    print_error(&error, 2);

    error = hipFree(d_x);
    print_error(&error, 2);

    error = hipFree(d_y);
    print_error(&error, 2);

    error = hipEventDestroy(start);
    print_error(&error, 2);

    error = hipEventDestroy(stop);
    print_error(&error, 2);

    return milliseconds / 1000.0;
}

float invoke_kernel_3(HLL_matrix *hll_matrix, double *x, double *z, int num_threads)
{
    int block_num = (hll_matrix->N * 32 + num_threads - 1) / num_threads;

    double *d_data, *d_x, *d_y;
    int *d_col_index, *d_max_nzr, *d_offsets;
    hipEvent_t start, stop;
    hipError_t error;

    // Allocazione memoria GPU
    hipMalloc(&d_data, hll_matrix->data_num * sizeof(double));
    hipMalloc(&d_col_index, hll_matrix->data_num * sizeof(int));
    hipMalloc(&d_max_nzr, hll_matrix->hacks_num * sizeof(int));
    hipMalloc(&d_offsets, hll_matrix->offsets_num * sizeof(int));
    hipMalloc(&d_x, hll_matrix->N * sizeof(double));
    hipMalloc(&d_y, hll_matrix->M * sizeof(double));

    // Copia dati host → device
    hipMemcpy(d_data, hll_matrix->data, hll_matrix->data_num * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_col_index, hll_matrix->col_index, hll_matrix->data_num * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_max_nzr, hll_matrix->max_nzr, hll_matrix->hacks_num * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_offsets, hll_matrix->offsets, hll_matrix->offsets_num * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_x, x, hll_matrix->N * sizeof(double), hipMemcpyHostToDevice);

    // Creazione e avvio eventi per misurare il tempo
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // Invocazione kernel con shared memory
    int shared_mem_size = hll_matrix->hack_size * sizeof(double) + hll_matrix->hack_size * sizeof(int);
    cuda_hll_kernel_v3<<<block_num, num_threads, shared_mem_size>>>(d_y, hll_matrix->hack_size, hll_matrix->hacks_num,
                                                                    d_data, d_offsets, d_col_index,
                                                                    d_max_nzr, d_x, hll_matrix->M);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // Misurazione del tempo di esecuzione
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    // Copia dei risultati da device → host
    hipMemcpy(z, d_y, hll_matrix->M * sizeof(double), hipMemcpyDeviceToHost);

    // Liberazione memoria
    hipFree(d_data);
    hipFree(d_col_index);
    hipFree(d_max_nzr);
    hipFree(d_offsets);
    hipFree(d_x);
    hipFree(d_y);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return milliseconds / 1000.0;
}