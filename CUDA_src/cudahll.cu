#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <string.h>
#include <errno.h>
#include <dirent.h>

#include "../CUDA_include/cudahll.h"
#include "../src/data_structures/hll_matrix.h"
#include "../src/data_structures/performance.h"
#include "../CUDA_include/hll/cuda_hll_kernel_v1.cuh"
#include "../CUDA_include/hll/cuda_hll_kernel_v2.cuh"
#include "../CUDA_include/hll/cuda_hll_kernel_v3.cuh"
#include "../CUDA_include/hll/cuda_hll_kernel_v4.cuh"

#define WARP_SIZE 32

void print_error(hipError_t *error, int kernel_index)
{
    if (*error != hipSuccess)
    {
        printf("Error occour in invoke kernel %d\nError: %s\n", kernel_index, hipGetErrorString(*error));
        exit(EXIT_FAILURE);
    }
}

float invoke_kernel_1(HLL_matrix *hll_matrix, double *x, double *z, int num_threads)
{
    dim3 block_dim(WARP_SIZE, num_threads);
    dim3 grid_dim((hll_matrix->N + block_dim.y - 1) / block_dim.y);

    double *d_data, *d_x, *d_y;
    int *d_col_index, *d_max_nzr, *d_offsets;
    hipEvent_t start, stop;
    hipError_t error;

    error = hipMalloc(&d_data, hll_matrix->data_num * sizeof(double));
    print_error(&error, 1);

    error = hipMalloc(&d_col_index, hll_matrix->data_num * sizeof(int));
    print_error(&error, 1);

    error = hipMalloc(&d_max_nzr, hll_matrix->hacks_num * sizeof(int));
    print_error(&error, 1);

    error = hipMalloc(&d_offsets, hll_matrix->offsets_num * sizeof(int));
    print_error(&error, 1);

    error = hipMalloc(&d_x, hll_matrix->N * sizeof(double));
    print_error(&error, 1);

    error = hipMalloc(&d_y, hll_matrix->M * sizeof(double));
    print_error(&error, 1);

    error = hipMemcpy(d_data, hll_matrix->data, hll_matrix->data_num * sizeof(double), hipMemcpyHostToDevice);
    print_error(&error, 1);

    error = hipMemcpy(d_col_index, hll_matrix->col_index, hll_matrix->data_num * sizeof(int), hipMemcpyHostToDevice);
    print_error(&error, 1);

    error = hipMemcpy(d_max_nzr, hll_matrix->max_nzr, hll_matrix->hacks_num * sizeof(int), hipMemcpyHostToDevice);
    print_error(&error, 1);

    error = hipMemcpy(d_offsets, hll_matrix->offsets, hll_matrix->offsets_num * sizeof(int), hipMemcpyHostToDevice);
    print_error(&error, 1);

    error = hipMemcpy(d_x, x, hll_matrix->N * sizeof(double), hipMemcpyHostToDevice);
    print_error(&error, 1);

    error = hipEventCreate(&start);
    print_error(&error, 1);

    error = hipEventCreate(&stop);
    print_error(&error, 1);

    error = hipEventRecord(start);
    print_error(&error, 1);

    cuda_hll_kernel_v1<<<grid_dim, num_threads>>>(d_y, hll_matrix->hack_size, hll_matrix->hacks_num, d_data, d_offsets, d_col_index,
                                                  d_max_nzr, d_x, hll_matrix->M);

    error = hipEventRecord(stop);
    print_error(&error, 1);

    error = hipEventSynchronize(stop);
    print_error(&error, 1);

    float milliseconds = 0;
    error = hipEventElapsedTime(&milliseconds, start, stop);
    print_error(&error, 1);

    error = hipMemcpy(z, d_y, hll_matrix->M * sizeof(double), hipMemcpyDeviceToHost);
    print_error(&error, 1);

    error = hipFree(d_data);
    print_error(&error, 1);

    error = hipFree(d_col_index);
    print_error(&error, 1);

    error = hipFree(d_max_nzr);
    print_error(&error, 1);

    error = hipFree(d_offsets);
    print_error(&error, 1);

    error = hipFree(d_x);
    print_error(&error, 1);

    error = hipFree(d_y);
    print_error(&error, 1);

    error = hipEventDestroy(start);
    print_error(&error, 1);

    error = hipEventDestroy(stop);
    print_error(&error, 1);

    return milliseconds / 1000.0;
}

float invoke_kernel_2(HLL_matrix *hll_matrix, double *x, double *z, int num_threads)
{
    dim3 block_dim(WARP_SIZE, num_threads);
    dim3 grid_dim((hll_matrix->N + block_dim.y - 1) / block_dim.y);

    double *d_data, *d_x, *d_y;
    int *d_col_index, *d_max_nzr, *d_offsets;
    hipEvent_t start, stop;
    hipError_t error;

    error = hipMalloc(&d_data, hll_matrix->data_num * sizeof(double));
    print_error(&error, 2);

    error = hipMalloc(&d_col_index, hll_matrix->data_num * sizeof(int));
    print_error(&error, 2);

    error = hipMalloc(&d_max_nzr, hll_matrix->hacks_num * sizeof(int));
    print_error(&error, 2);

    error = hipMalloc(&d_offsets, hll_matrix->offsets_num * sizeof(int));
    print_error(&error, 2);

    error = hipMalloc(&d_x, hll_matrix->N * sizeof(double));
    print_error(&error, 2);

    error = hipMalloc(&d_y, hll_matrix->N * sizeof(double));
    print_error(&error, 2);

    error = hipMemcpy(d_data, hll_matrix->data, hll_matrix->data_num * sizeof(double), hipMemcpyHostToDevice);
    print_error(&error, 2);

    error = hipMemcpy(d_col_index, hll_matrix->col_index, hll_matrix->data_num * sizeof(int), hipMemcpyHostToDevice);
    print_error(&error, 2);

    error = hipMemcpy(d_max_nzr, hll_matrix->max_nzr, hll_matrix->hacks_num * sizeof(int), hipMemcpyHostToDevice);
    print_error(&error, 2);

    error = hipMemcpy(d_offsets, hll_matrix->offsets, hll_matrix->offsets_num * sizeof(int), hipMemcpyHostToDevice);
    print_error(&error, 2);

    error = hipMemcpy(d_x, x, hll_matrix->N * sizeof(double), hipMemcpyHostToDevice);
    print_error(&error, 2);

    error = hipEventCreate(&start);
    print_error(&error, 2);

    error = hipEventCreate(&stop);
    print_error(&error, 2);

    error = hipEventRecord(start);
    print_error(&error, 2);

    cuda_hll_kernel_v2<<<grid_dim, num_threads>>>(d_y, hll_matrix->hack_size, hll_matrix->hacks_num, d_data, d_offsets,
                                                  d_col_index, d_max_nzr, d_x, hll_matrix->M);

    error = hipEventRecord(stop);
    print_error(&error, 2);

    error = hipEventSynchronize(stop);
    print_error(&error, 2);

    float milliseconds = 0;
    error = hipEventElapsedTime(&milliseconds, start, stop);
    print_error(&error, 2);

    error = hipMemcpy(z, d_y, hll_matrix->M * sizeof(double), hipMemcpyDeviceToHost);
    print_error(&error, 2);

    error = hipFree(d_data);
    print_error(&error, 2);

    error = hipFree(d_col_index);
    print_error(&error, 2);

    error = hipFree(d_max_nzr);
    print_error(&error, 2);

    error = hipFree(d_offsets);
    print_error(&error, 2);

    error = hipFree(d_x);
    print_error(&error, 2);

    error = hipFree(d_y);
    print_error(&error, 2);

    error = hipEventDestroy(start);
    print_error(&error, 2);

    error = hipEventDestroy(stop);
    print_error(&error, 2);

    return milliseconds / 1000.0;
}

float invoke_kernel_3(HLL_matrix *hll_matrix, double *x, double *z, int num_threads)
{
    dim3 block_dim(WARP_SIZE, num_threads);
    dim3 grid_dim((hll_matrix->N + block_dim.y - 1) / block_dim.y);

    double *d_data, *d_x, *d_y;
    int *d_col_index, *d_max_nzr, *d_offsets;
    hipEvent_t start, stop;
    hipError_t error;

    error = hipMalloc(&d_data, hll_matrix->data_num * sizeof(double));
    print_error(&error, 3);

    error = hipMalloc(&d_col_index, hll_matrix->data_num * sizeof(int));
    print_error(&error, 3);

    error = hipMalloc(&d_max_nzr, hll_matrix->hacks_num * sizeof(int));
    print_error(&error, 3);

    error = hipMalloc(&d_offsets, hll_matrix->offsets_num * sizeof(int));
    print_error(&error, 3);

    error = hipMalloc(&d_x, hll_matrix->N * sizeof(double));
    print_error(&error, 3);

    error = hipMalloc(&d_y, hll_matrix->N * sizeof(double));
    print_error(&error, 3);

    error = hipMemcpy(d_data, hll_matrix->data, hll_matrix->data_num * sizeof(double), hipMemcpyHostToDevice);
    print_error(&error, 3);

    error = hipMemcpy(d_col_index, hll_matrix->col_index, hll_matrix->data_num * sizeof(int), hipMemcpyHostToDevice);
    print_error(&error, 3);

    error = hipMemcpy(d_max_nzr, hll_matrix->max_nzr, hll_matrix->hacks_num * sizeof(int), hipMemcpyHostToDevice);
    print_error(&error, 3);

    error = hipMemcpy(d_offsets, hll_matrix->offsets, hll_matrix->offsets_num * sizeof(int), hipMemcpyHostToDevice);
    print_error(&error, 3);

    error = hipMemcpy(d_x, x, hll_matrix->N * sizeof(double), hipMemcpyHostToDevice);
    print_error(&error, 3);

    error = hipEventCreate(&start);
    print_error(&error, 3);

    error = hipEventCreate(&stop);
    print_error(&error, 3);

    error = hipEventRecord(start);
    print_error(&error, 3);

    int shared_mem_size = 1024 * sizeof(double);
    cuda_hll_kernel_v3<<<grid_dim, num_threads, shared_mem_size>>>(d_y, hll_matrix->hack_size, hll_matrix->hacks_num,
                                                                   d_data, d_offsets, d_col_index, d_max_nzr, d_x,
                                                                   hll_matrix->M);

    error = hipEventRecord(stop);
    print_error(&error, 3);

    error = hipEventSynchronize(stop);
    print_error(&error, 3);

    float milliseconds = 0;
    error = hipEventElapsedTime(&milliseconds, start, stop);
    print_error(&error, 3);

    error = hipMemcpy(z, d_y, hll_matrix->N * sizeof(double), hipMemcpyDeviceToHost);
    print_error(&error, 3);

    error = hipFree(d_data);
    print_error(&error, 3);

    error = hipFree(d_col_index);
    print_error(&error, 3);

    error = hipFree(d_max_nzr);
    print_error(&error, 3);

    error = hipFree(d_offsets);
    print_error(&error, 3);

    error = hipFree(d_x);
    print_error(&error, 3);

    error = hipFree(d_y);
    print_error(&error, 3);

    error = hipEventDestroy(start);
    print_error(&error, 3);

    error = hipEventDestroy(stop);
    print_error(&error, 3);

    return milliseconds / 1000.0;
}

float invoke_kernel_4(HLL_matrix *hll_matrix, double *x, double *z, int num_threads)
{
    int block_num = (hll_matrix->N * 32) / num_threads;

    double *d_data, *d_x, *d_y;
    int *d_col_index, *d_max_nzr, *d_offsets;
    hipEvent_t start, stop;
    hipError_t error;

    error = hipMalloc(&d_data, hll_matrix->data_num * sizeof(double));
    print_error(&error, 4);

    error = hipMalloc(&d_col_index, hll_matrix->data_num * sizeof(int));
    print_error(&error, 4);

    error = hipMalloc(&d_max_nzr, hll_matrix->hacks_num * sizeof(int));
    print_error(&error, 4);

    error = hipMalloc(&d_offsets, hll_matrix->offsets_num * sizeof(int));
    print_error(&error, 4);

    error = hipMalloc(&d_x, hll_matrix->N * sizeof(double));
    print_error(&error, 4);

    error = hipMalloc(&d_y, hll_matrix->N * sizeof(double));
    print_error(&error, 4);

    error = hipMemcpy(d_data, hll_matrix->data, hll_matrix->data_num * sizeof(double), hipMemcpyHostToDevice);
    print_error(&error, 4);

    error = hipMemcpy(d_col_index, hll_matrix->col_index, hll_matrix->data_num * sizeof(int), hipMemcpyHostToDevice);
    print_error(&error, 4);

    error = hipMemcpy(d_max_nzr, hll_matrix->max_nzr, hll_matrix->hacks_num * sizeof(int), hipMemcpyHostToDevice);
    print_error(&error, 4);

    error = hipMemcpy(d_offsets, hll_matrix->offsets, hll_matrix->offsets_num * sizeof(int), hipMemcpyHostToDevice);
    print_error(&error, 4);

    error = hipMemcpy(d_x, x, hll_matrix->N * sizeof(double), hipMemcpyHostToDevice);
    print_error(&error, 4);

    error = hipEventCreate(&start);
    print_error(&error, 4);

    error = hipEventCreate(&stop);
    print_error(&error, 4);

    error = hipEventRecord(start);
    print_error(&error, 4);

    cuda_hll_kernel_v4<<<block_num, num_threads>>>(d_y, hll_matrix->hack_size, hll_matrix->hacks_num,
                                                   d_data, d_offsets, d_col_index, d_max_nzr, d_x,
                                                   hll_matrix->M);

    error = hipEventRecord(stop);
    print_error(&error, 4);

    error = hipEventSynchronize(stop);
    print_error(&error, 4);

    float milliseconds = 0;
    error = hipEventElapsedTime(&milliseconds, start, stop);
    print_error(&error, 4);

    error = hipMemcpy(z, d_y, hll_matrix->N * sizeof(double), hipMemcpyDeviceToHost);
    print_error(&error, 4);

    error = hipFree(d_data);
    print_error(&error, 4);

    error = hipFree(d_col_index);
    print_error(&error, 4);

    error = hipFree(d_max_nzr);
    print_error(&error, 4);

    error = hipFree(d_offsets);
    print_error(&error, 4);

    error = hipFree(d_x);
    print_error(&error, 4);

    error = hipFree(d_y);
    print_error(&error, 4);

    error = hipEventDestroy(start);
    print_error(&error, 4);

    error = hipEventDestroy(stop);
    print_error(&error, 4);

    return milliseconds / 1000.0;
}