#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <iostream>

#include "../CUDA_include/cudacsr.h"
#include "../CUDA_include/cuda_csr_kernel_v1.cuh"

// qua alloca strutture dati per le chiamate a kernel (partizione del carico di ogni warp)

float invoke_kernel_v1(CSR_matrix *csr_matrix, double *x, double *z)
{
    double *d_x, *d_y;
    int *d_row, *d_col;
    double *d_val;
    hipMalloc(&d_x, csr_matrix->N * sizeof(double));
    hipMalloc(&d_y, csr_matrix->N * sizeof(double));
    hipMalloc(&d_row, (csr_matrix->N + 1) * sizeof(int));
    hipMalloc(&d_col, csr_matrix->non_zero_values * sizeof(int));
    hipMalloc(&d_val, csr_matrix->non_zero_values * sizeof(double));
    if (d_x == NULL || d_y == NULL || d_row == NULL || d_col == NULL || d_val == NULL)
    {
        printf("Errore nell'allocazione della memoria per il device\n");
        exit(EXIT_FAILURE);
    }

    hipMemcpy(d_x, x, csr_matrix->N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, csr_matrix->N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_row, csr_matrix->IRP, (csr_matrix->N + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_col, csr_matrix->JA, csr_matrix->non_zero_values * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_val, csr_matrix->AS, csr_matrix->non_zero_values * sizeof(double), hipMemcpyHostToDevice);

    int block_size = 256;
    int num_blocks = (csr_matrix->N + block_size - 1) / block_size;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    csr_matvec_warps_per_row<<<num_blocks, block_size>>>(csr->N, d_row, d_col, d_val, d_x, d_y);
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    hipMemcpy(z, d_y, csr_matrix->M * sizeof(double), hipMemcpyDeviceToHost);

    printf("Milliseconds = %.16lf\n", milliseconds);

    time = milliseconds / 1000.0;

    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_row);
    hipFree(d_col);
    hipFree(d_val);

    return time;
}
