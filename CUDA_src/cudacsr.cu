#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <iostream>

#include "../CUDA_include/cudacsr.h"
#include "../CUDA_include/cuda_csr_kernel_v1.cuh"

// qua alloca strutture dati per le chiamate a kernel (partizione del carico di ogni warp)

float invoke_kernel_csr_1(CSR_matrix *csr_matrix, double *x, double *z)
{
    CSR_matrix d_A;
    double *d_x, *d_y;
    float time;

    hipMalloc(&d_A.IRP, (csr_matrix->M + 1) * sizeof(int));
    hipMalloc(&d_A.JA, csr_matrix->non_zero_values * sizeof(int));
    hipMalloc(&d_A.AS, csr_matrix->non_zero_values * sizeof(double));

    hipMalloc(&d_x, csr_matrix->N * sizeof(double));
    hipMalloc(&d_y, csr_matrix->M * sizeof(double));

    hipMemcpy(d_A.IRP, csr_matrix->IRP, (csr_matrix->M + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_A.JA, csr_matrix->JA, csr_matrix->non_zero_values * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_A.AS, csr_matrix->AS, csr_matrix->non_zero_values * sizeof(double), hipMemcpyHostToDevice);

    hipMemcpy(d_x, x, csr_matrix->N * sizeof(double), hipMemcpyHostToDevice);
    hipMemset(d_y, 0, csr_matrix->M * sizeof(double));

    int blockSize = 256;
    int gridSize = (csr_matrix->M + blockSize - 1) / blockSize;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    csr_matvec_kernel<<<gridSize, blockSize>>>(d_A, d_x, d_y);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    hipMemcpy(z, d_y, csr_matrix->M * sizeof(double), hipMemcpyDeviceToHost);
    printf("Milliseconds = %.16lf\n", milliseconds);

    time = milliseconds / 1000.0;

    hipFree(d_A.IRP);
    hipFree(d_A.JA);
    hipFree(d_A.AS);
    hipFree(d_x);
    hipFree(d_y);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return time;
}
