#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <iostream>

#include "../CUDA_include/cudacsr.h"
#include "../CUDA_include/csr/cuda_csr_kernel_v1.cuh"
#include "../CUDA_include/csr/cuda_csr_kernel_v2.cuh"

// qua alloca strutture dati per le chiamate a kernel (partizione del carico di ogni warp)

/*Kernel 1*/
float invoke_kernel_csr_1(CSR_matrix *csr_matrix, double *x, double *z)
{
    CSR_matrix d_A;
    double *d_x, *d_y;
    float time;

    hipMalloc(&d_A.IRP, (csr_matrix->M + 1) * sizeof(int));
    hipMalloc(&d_A.JA, csr_matrix->non_zero_values * sizeof(int));
    hipMalloc(&d_A.AS, csr_matrix->non_zero_values * sizeof(double));
    hipMalloc(&d_x, csr_matrix->N * sizeof(double));
    hipMalloc(&d_y, csr_matrix->M * sizeof(double));

    hipMemcpy(d_A.IRP, csr_matrix->IRP, (csr_matrix->M + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_A.JA, csr_matrix->JA, csr_matrix->non_zero_values * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_A.AS, csr_matrix->AS, csr_matrix->non_zero_values * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_x, x, csr_matrix->N * sizeof(double), hipMemcpyHostToDevice);
    hipMemset(d_y, 0, csr_matrix->M * sizeof(double));

    int blockSize = 8;
    // 256  -> 16
    // 32   -> 22
    // 16   -> 41.8
    // 8    -> 53.41
    // 10   -> 51.96
    // 11   -> 50.85
    // 12   -> 51.13
    // 64   -> 15.40
    // 128  -> 15.61
    // 512  -> 15.38
    // 4    -> 35.82
    int gridSize = (csr_matrix->M + blockSize - 1) / blockSize;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    csr_matvec_kernel<<<gridSize, blockSize>>>(d_A, d_x, d_y);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    hipMemcpy(z, d_y, csr_matrix->M * sizeof(double), hipMemcpyDeviceToHost);
    printf("Milliseconds = %.16lf\n", milliseconds);

    time = milliseconds / 1000.0;

    hipFree(d_A.IRP);
    hipFree(d_A.JA);
    hipFree(d_A.AS);
    hipFree(d_x);
    hipFree(d_y);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return time;
}

/*Kernel 2*/
float invoke_kernel_csr_2(CSR_matrix *csr_matrix, double *x, double *z)
{
    hipEvent_t start, stop;
    float elapsedTime;
    double *d_x, *d_y;

    CSR_matrix d_mat;
    hipMalloc(&d_mat.IRP, (csr_matrix->M + 1) * sizeof(int));
    hipMalloc(&d_mat.JA, csr_matrix->non_zero_values * sizeof(int));
    hipMalloc(&d_mat.AS, csr_matrix->non_zero_values * sizeof(double));
    hipMalloc(&d_x, csr_matrix->N * sizeof(double));
    hipMalloc(&d_y, csr_matrix->M * sizeof(double));

    hipMemcpy(d_mat.IRP, csr_matrix->IRP, (csr_matrix->M + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_mat.JA, csr_matrix->JA, csr_matrix->non_zero_values * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_mat.AS, csr_matrix->AS, csr_matrix->non_zero_values * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_x, x, csr_matrix->N * sizeof(double), hipMemcpyHostToDevice);
    hipMemset(d_y, 0, csr_matrix->M * sizeof(double));

    // Configurazione blocchi e griglia kernel
    dim3 blockDim(WARP_SIZE, 512);
    // 256  -> mala noticia
    // 32   -> mala noticia
    // 16   -> mala noticia
    // 8    -> mala noticia
    // 64   -> mala noticia
    // 128  -> mala noticia
    // 512  -> mala noticia
    // 5    -> mala noticia
    // 4    -> 50.82
    // 3    -> 49.62

    dim3 gridDim((csr_matrix->M + blockDim.y - 1) / blockDim.y);
    printf("blockDim.y = %d\n", blockDim.y);
    printf("gridDim = %d\n", blockDim.y);

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    size_t sharedMemSize = blockDim.y * WARP_SIZE * sizeof(double);
    csr_matvec_warp_shmem<<<gridDim, blockDim, sharedMemSize>>>(d_mat, d_x, d_y);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);

    printf("Tempo di esecuzione: %.10f ms\n", elapsedTime);

    // Risultato da GPU a CPU
    hipMemcpy(z, d_y, csr_matrix->M * sizeof(double), hipMemcpyDeviceToHost);

    // Libera memoria sulla GPU
    hipFree(d_mat.IRP);
    hipFree(d_mat.JA);
    hipFree(d_mat.AS);
    hipFree(d_x);
    hipFree(d_y);

    return elapsedTime / 1000;
}

// float invoke_kernel_csr_3(CSR_matrix *csr_matrix, double *x, double *z)
// {}