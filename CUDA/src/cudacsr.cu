#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <string.h>
#include <errno.h>
#include <dirent.h>

#include "../CUDA_include/cudacsr.h"
#include "../src/data_structures/csr_matrix.h"
#include "../src/data_structures/performance.h"
#include "../CUDA_include/cuda_csr_kernel_v1.cuh"

// qua alloca strutture dati per le chiamate a kernel (partizione del carico di ogni warp)

double call_kernel_v1(CSR_matrix *csr, double *x, double *y)
{
    double *d_x, *d_y;
    int *d_row, *d_col;
    double *d_val;
    double *d_res;
    double res;
    hipMalloc(&d_x, csr->N * sizeof(double));
    hipMalloc(&d_y, csr->N * sizeof(double));
    hipMalloc(&d_row, (csr->N + 1) * sizeof(int));
    hipMalloc(&d_col, csr->non_zero_values * sizeof(int));
    hipMalloc(&d_val, csr->non_zero_values * sizeof(double));
    hipMalloc(&d_res, sizeof(double));
    if (d_x == NULL || d_y == NULL || d_row == NULL || d_col == NULL || d_val == NULL || d_res == NULL)
    {
        printf("Errore nell'allocazione della memoria per il device\n");
        exit(EXIT_FAILURE);
    }

    hipMemcpy(d_x, x, csr->N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, csr->N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_row, csr->IRP, (csr->N + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_col, csr->JA, csr->non_zero_values * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_val, csr->AS, csr->non_zero_values * sizeof(double), hipMemcpyHostToDevice);

    int block_size = 256;
    int num_blocks = (csr->N + block_size - 1) / block_size;
    csr_matvec_warps_per_row<<<num_blocks, block_size>>>(csr->N, d_row, d_col, d_val, d_x, d_y, d_res);
    // mi permette di sincronizzare il device con l'host
    hipDeviceSynchronize();
    // copia risultato da device
    hipMemcpy(&res, d_res, sizeof(double), hipMemcpyDeviceToHost);

    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_row);
    hipFree(d_col);
    hipFree(d_val);
    hipFree(d_res);

    return res;
}