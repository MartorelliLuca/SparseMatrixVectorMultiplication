#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include "../headers/cudacsr.h"
#include "../kernel/csr/cudakernel1.cuh"

// qua alloca strutture dati per le chiamate a kernel (partizione del carico di ogni warp)

double call_kernel_v1(CSRMatrix *csr, double *x, double *y)
{

    double *d_x, *d_y;
    int *d_row, *d_col;
    double *d_val;
    double *d_res;
    double res;
    hipMalloc(&d_x, csr->n * sizeof(double));
    hipMalloc(&d_y, csr->n * sizeof(double));
    hipMalloc(&d_row, (csr->n + 1) * sizeof(int));
    hipMalloc(&d_col, csr->nnz * sizeof(int));
    hipMalloc(&d_val, csr->nnz * sizeof(double));
    hipMalloc(&d_res, sizeof(double));
    if (d_x == NULL || d_y == NULL || d_row == NULL || d_col == NULL || d_val == NULL || d_res == NULL)
    {
        printf("Errore nell'allocazione della memoria per il device\n");
        exit(EXIT_FAILURE);
    }

    hipMemcpy(d_x, x, csr->n * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, csr->n * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_row, csr->row, (csr->n + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_col, csr->col, csr->nnz * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_val, csr->val, csr->nnz * sizeof(double), hipMemcpyHostToDevice);

    int block_size = 256;
    int num_blocks = (csr->n + block_size - 1) / block_size;
    csr_matvec_warps_per_row<<<num_blocks, block_size>>>(csr->n, d_row, d_col, d_val, d_x, d_y, d_res);
    // mi permette di sincronizzare il device con l'host
    hipDeviceSynchronize();
    // copia risultato da device
    hipMemcpy(&res, d_res, sizeof(double), hipMemcpyDeviceToHost);

    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_row);
    hipFree(d_col);
    hipFree(d_val);
    hipFree(d_res);

    return res;
}