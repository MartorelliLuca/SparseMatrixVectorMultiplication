#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <string.h>
#include <errno.h>
#include <dirent.h>

#include "../../src/data_structures/hll_matrix.h"
#include "../../src/data_structures/performance.h"
#include "../kernel/hll/cuda_hll_kernel_v1.cuh"

double invoke_kernel_1(HLL_matrix *hll_matrix, double *x, double *z)
{
    hipError_t error;
    int *d_offsets, *d_col_index;
    double *d_data, *d_x, *d_y;
    float time = 0.0;

    // Eventi CUDA per la misurazione del tempo
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Allocazione memoria GPU con controlli errori migliorati
    if (hipMalloc((void **)&d_offsets, hll_matrix->offsets_num * sizeof(int)) != hipSuccess ||
        hipMalloc((void **)&d_col_index, hll_matrix->data_num * sizeof(int)) != hipSuccess ||
        hipMalloc((void **)&d_data, hll_matrix->data_num * sizeof(double)) != hipSuccess ||
        hipMalloc((void **)&d_x, hll_matrix->N * sizeof(double)) != hipSuccess ||
        hipMalloc((void **)&d_y, hll_matrix->M * sizeof(double)) != hipSuccess)
    {
        error = hipGetLastError();
        printf("Error during hipMalloc!\nError code: %d\n", error);
        exit(EXIT_FAILURE);
    }

    // Copia dati da host a device
    hipMemcpy(d_offsets, hll_matrix->offsets, hll_matrix->offsets_num * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_col_index, hll_matrix->col_index, hll_matrix->data_num * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_data, hll_matrix->data, hll_matrix->data_num * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_x, x, hll_matrix->N * sizeof(double), hipMemcpyHostToDevice);

    // Configurazione kernel
    int blockSize = 32;
    int numBlocks = (hll_matrix->M + blockSize - 1) / blockSize;

    // Avvio misurazione tempo
    hipEventRecord(start);

    // Esecuzione del kernel CUDA
    hll_kernel_v1<<<numBlocks, blockSize>>>(d_offsets, d_col_index, d_data, d_x, d_y, hll_matrix->M);

    // Sincronizzazione e controllo errori
    hipDeviceSynchronize();
    error = hipGetLastError();
    if (error != hipSuccess)
    {
        printf("Error during kernel execution!\nError code: %d\n", error);
        exit(EXIT_FAILURE);
    }

    // Stop misurazione tempo
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);

    // Copia risultati da device a host
    hipMemcpy(z, d_y, hll_matrix->M * sizeof(double), hipMemcpyDeviceToHost);

    // Libera memoria GPU
    hipFree(d_offsets);
    hipFree(d_col_index);
    hipFree(d_data);
    hipFree(d_x);
    hipFree(d_y);

    // Distruggi eventi CUDA
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return time / 1000.0; // Convertiamo da millisecondi a secondi
}
