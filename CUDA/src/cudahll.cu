#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <string.h>
#include <errno.h>
#include <dirent.h>

#include "../../src/data_structures/hll_matrix.h"
#include "../../src/data_structures/performance.h"
#include "../kernel/hll/cuda_hll_kernel_v1.cuh"

double prepare_kernel_v1(HLL_matrix *hll_matrix, double *x, double *z)
{
    hipError_t error;
    int *d_offsets, *d_col_index;
    double *d_data, *d_x, *d_y;
    double time = 0.0;

    // Create memory on GPU
    if (hipMalloc((void **)&d_offsets, hll_matrix->offsets_num * sizeof(int)) != hipSuccess)
    {
        error = hipGetLastError();
        printf("Error occour in hipMalloc for d_offset!\nError code: %d\n", error);
        exit(EXIT_FAILURE);
    }

    if (hipMalloc((void **)&d_col_index, hll_matrix->data_num * sizeof(int)) != hipSuccess)
    {
        error = hipGetLastError();
        printf("Error occour in hipMalloc for d_col_index!\nError code: %d\n", error);
        exit(EXIT_FAILURE);
    }

    if (hipMalloc((void **)&d_data, hll_matrix->data_num * sizeof(double)) != hipSuccess)
    {
        error = hipGetLastError();
        printf("Error occour in hipMalloc for d_data!\nError code: %d\n", error);
        exit(EXIT_FAILURE);
    }

    if (hipMalloc((void **)&d_x, hll_matrix->N * sizeof(double)) != hipSuccess)
    {
        error = hipGetLastError();
        printf("Error occour in hipMalloc for d_x!\nError code: %d\n", error);
        exit(EXIT_FAILURE);
    }

    if (hipMalloc((void **)&d_y, hll_matrix->M * sizeof(double)) != hipSuccess)
    {
        error = hipGetLastError();
        printf("Error occour in hipMalloc for d_y!\nError code: %d\n", error);
        exit(EXIT_FAILURE);
    }

    // Copy data from host to device
    if (hipMemcpy(d_offsets, hll_matrix->offsets, hll_matrix->offsets_num * sizeof(int), hipMemcpyHostToDevice) != hipSuccess)
    {
        error = hipGetLastError();
        printf("Error occour in hipMemcpy for d_offset!\nError code: %d\n", error);
        exit(EXIT_FAILURE);
    }

    if (hipMemcpy(d_col_index, hll_matrix->col_index, hll_matrix->data_num * sizeof(int), hipMemcpyHostToDevice) != hipSuccess)
    {
        error = hipGetLastError();
        printf("Error occour in hipMemcpy for d_col_index!\nError code: %d\n", error);
        exit(EXIT_FAILURE);
    }

    if (hipMemcpy(d_data, hll_matrix->data, hll_matrix->data_num * sizeof(double), hipMemcpyHostToDevice) != hipSuccess)
    {
        error = hipGetLastError();
        printf("Error occour in hipMemcpy for d_data!\nError code: %d\n", error);
        exit(EXIT_FAILURE);
    }

    if (hipMemcpy(d_x, x, hll_matrix->N * sizeof(double), hipMemcpyHostToDevice) != hipSuccess)
    {
        error = hipGetLastError();
        printf("Error occour in hipMalloc for d_x!\nError code: %d\n", error);
        exit(EXIT_FAILURE);
    }

    // Kernel CUDA Configuration
    int blockSize = 32;
    int numBlocks = (hll_matrix->M + blockSize - 1) / blockSize;

    // Run Kernel CUDA
    hll_kernel_v1<<<numBlocks, blockSize>>>(d_offsets, d_col_index, d_data, d_x, d_y, hll_matrix->M);
    hipDeviceSynchronize();

    // Copy of result
    hipMemcpy(z, d_y, hll_matrix->M * sizeof(double), hipMemcpyDeviceToHost);

    // Free GPU memory
    hipFree(d_offsets);
    hipFree(d_col_index);
    hipFree(d_data);
    hipFree(d_x);
    hipFree(d_y);

    return time;
}